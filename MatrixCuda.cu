
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void matrixMul(const float *A, const float *B, float *C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N) {
        float sum = 0;
        for (int i = 0; i < K; i++) {
            sum += A[row * K + i] * B[i * N + col];
        }
        C[row * N + col] = sum;
    }
}
void printMatrix(float *matrix, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%f ", matrix[i * cols + j]);
        }
        printf("\n");
    }
}

int main() {
    // Set matrix dimensions
    int M = 5000;
    int N = 6000;
    int K = 400;

    // Allocate memory on the host for input and output matrices
    float *A, *B, *C;
    A = (float *)malloc(M * K * sizeof(float));
    B = (float *)malloc(K * N * sizeof(float));
    C = (float *)malloc(M * N * sizeof(float));

    // Initialize input matrices with random values
    for (int i = 0; i < M * K; i++) A[i] = rand() / (float)RAND_MAX;
    for (int i = 0; i < K * N; i++) B[i] = rand() / (float)RAND_MAX;

    // printf("MatrixA:\n");
    // printMatrix(A, M, K);

    // printf("MatrixB:\n");
    // printMatrix(B, M, K);

    // Allocate memory on the device for input and output matrices
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, M * K * sizeof(float));
    hipMalloc((void **)&d_B, K * N * sizeof(float));
    hipMalloc((void **)&d_C, M * N * sizeof(float));

    // Copy input matrices from host to device memory
    hipMemcpy(d_A, A, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, K * N * sizeof(float), hipMemcpyHostToDevice);

      // Create and record events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // Set block and grid dimensions
    dim3 blockDim(16, 16);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (M + blockDim.y - 1) / blockDim.y);

    // Launch kernel on the device
    matrixMul<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, N, K);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    

    // Copy output matrix from device to host memory
    hipMemcpy(C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);

    // printf("Matrix C:\n");
    // printMatrix(C, M, N);

    printf("Execution Time: %f ms\n", elapsedTime);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(A);
    free(B);
    free(C);

    return 0;
}
